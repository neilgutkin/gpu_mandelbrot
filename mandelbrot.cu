#include "hip/hip_runtime.h"
// Neil Gutkin
// 09/29/21

/*

To compile:
nvcc -O3 -o mandelbrot mandelbrot.cu png_util.c -I. -lm -lpng

To run:
./mandelbrot

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// include as a C file (no name mangling)
extern "C" {
  #include "png_util.h"
}

/*
z <- z^2 + c
Perform the above iteration for each complex value c and count
how many iterations it takes before the magnitude of the complex number
z satisfies |z|<4
*/
__global__ void mandelbrotKernel(const int NRe, 
		const int NIm, 
		const float minRe,
		const float minIm,
		const float dRe, 
		const float dIm,
		float * h_count){

  // orient thread
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int dx = blockDim.x;
  
  int ty = threadIdx.y;
  int by = blockIdx.y;
  int dy = blockDim.y;
  
  // establish which entry this thread is responsible for
  int column = tx + bx*dx;
  int row    = ty + by*dy;

  if (row < NIm && column < NRe) { // bounds check
      float cRe = minRe + column*dRe;
      float cIm = minIm + row*dIm;

      float zRe = 0;
      float zIm = 0;
      
      int Nt = 200;
      int t, cnt=0;
      for(t=0;t<Nt;++t){
	
	// z = z^2 + c
	//   = (zRe + i*zIm)*(zRe + i*zIm) + (cRe + i*cIm)
	//   = zRe^2 - zIm^2 + 2*i*zIm*zRe + cRe + i*cIm
	      float zReTmp = zRe*zRe - zIm*zIm + cRe;
	      zIm = 2.f*zIm*zRe + cIm;
	      zRe = zReTmp;

	      cnt += (zRe*zRe+zIm*zIm<4.f);
      }

      h_count[column + row*NRe] = cnt;
  }

}


int main(int argc, char **argv){

  const int NRe = 4096;
  const int NIm = 4096;

  // box containing sample points 
  const float centRe = -1.2, centIm= -.2;
  const float diam  = 0.3;
  const float minRe = centRe-0.5*diam;
  const float remax = centRe+0.5*diam;
  const float minIm = centIm-0.5*diam;
  const float immax = centIm+0.5*diam;

  const float dRe = (remax-minRe)/(NRe-1.f);
  const float dIm = (immax-minIm)/(NIm-1.f);

  // allocate HOST array
  float *h_count = (float*) calloc(NRe*NIm, sizeof(float));

  // allocate DEVICE array
  float* c_count;
  hipMalloc(&c_count, NRe*NIm * sizeof(float));

  // establish threading dimensions
  dim3 B(16,16,1);
  dim3 G((NIm+16-1)/16, (NRe+16-1)/16, 1);

  // warm up kernel
  mandelbrotKernel <<< G , B >>> (NRe, NIm, minRe, minIm, dRe, dIm, c_count);

  // set up timing
  hipEvent_t tic, toc;
  hipEventCreate(&tic);
  hipEventCreate(&toc);

  // start timer
  hipDeviceSynchronize();
  hipEventRecord(tic);

  // call mandelbrot from here
  mandelbrotKernel <<< G , B >>> (NRe, NIm, minRe, minIm, dRe, dIm, c_count);

  // end timer
  hipEventRecord(toc);
  hipDeviceSynchronize();

  // get and print elapsed time
  float elapsed;
  hipEventElapsedTime(&elapsed, tic, toc);
  elapsed /= 1000;
  
  printf("elapsed time %f seconds\n", elapsed);

  // copy results from DEVICE to HOST
  hipMemcpy(h_count, c_count, NRe*NIm * sizeof(float), hipMemcpyDeviceToHost);

  // create the png
  FILE *png = fopen("mandelbrot.png", "w");
  write_hot_png(png, NRe, NIm, h_count, 0, 80);
  fclose(png);

}
